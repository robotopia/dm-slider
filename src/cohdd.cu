#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include "cudaErrorChecking.h"

#include <stdlib.h>
#include <stdio.h>

#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>

#include <cuda_gl_interop.h>

/**
 * Convert a VDIF buffer into an array of floats
 *
 * @param in              Pointer to the VDIF buffer
 * @param out             Pointer to the output buffer
 * @param frameSizeBytes  The number of bytes per VDIF frame
 *                        (including the header)
 * @param headerSizeBytes The number of bytes per VDIF frame header
 *
 * Each thread converts one complex sample (= 2 bytes of input)
 */
__global__ void cudaVDIFToFloatComplex_kernel( uint8_t *in, hipFloatComplex *out, int frameSizeBytes, int headerSizeBytes, int c )
{
    // The size of just the data part of the frame
    int dataSizeBytes = frameSizeBytes - headerSizeBytes;

    // It is assumed that `in` points to the first byte in a frameheader
    int i = threadIdx.x + blockIdx.x*blockDim.x; // Index of (non-header) data sample
    int p = i % Np; // 
    int c = 

    // Express the index in terms of bytes
    int i2 = i*sizeof(uint8_t)*2;

    // Get the frame number for this byte, and the idx within this frame
    int frame      = i2 / dataSizeBytes;
    int idxInFrame = i2 % dataSizeBytes;

    // Calculate the indices into the input and output arrays for this sample
    int in_idx  = frame*frameSizeBytes + (headerSizeBytes + idxInFrame);
    int out_idx = i;

    // Bring the sample to register memory
    uint8_t sample_x = in[in_idx];
    uint8_t sample_y = in[in_idx+1];

    // Turn it into a float and write it to global memory
    out[out_idx] = make_hipFloatComplex(
            ((float)sample_x)/256.0f - 0.5f,
            ((float)sample_y)/256.0f - 0.5f );
}

/**
 * Apply a phase ramp to complex data
 *
 * @param data          The data to which the phase ramp is applied (in-place)
 * @param radPerBin     The slope of the phase ramp (in radians per bin)
 * @param samplesPerBin The number of contiguous samples to be rotated by the
 *                      same amount
 */
__global__ void cudaApplyPhaseRamp_kernel( hipFloatComplex *data, float radPerBin, int samplesPerBin )
{
    // For this block/thread...
    int s = threadIdx.x + blockIdx.x*blockDim.x; // Get the (s)ample number
    int b = s / samplesPerBin;                   // Get the (b)in number

    // For each bin, calculate the phase rotation to be applied
    float rad = b * radPerBin;
    hipFloatComplex phase;
    sincosf( rad, &phase.y, &phase.x );

    // Apply the phase ramp (in-place)
    data[s] = hipCmulf( data[s], phase );
}

/**
 * Convert dual polarisation data to Stokes I
 *
 * @param data The data to be converted
 * @param stokesI The Stokes I output
 *
 * `data` is expected to be an array of *pairs* of complex numbers,
 * X,Y,X,Y,X,Y,...
 * from which the Stokes parameters are formed:
 *    I = |X|^2 + |Y|^2
 */
__global__ void cudaStokesI_kernel( hipFloatComplex *data, float *stokesI )
{
    // Let i represent the output sample index
    int i = threadIdx.x + blockIdx.x*blockDim.x;

    // Pull out the two polarisations
    hipFloatComplex X = data[2*i];
    hipFloatComplex Y = data[2*i + 1];

    // Calculate Stokes I
    stokesI[i] = X.x*X.x + X.y*X.y + Y.x*Y.x + Y.y*Y.y;
}

__global__
void cudaCreateImage_kernel( float *image, int width, int height )
{
    // Makes an image of pixels ranging from 0.0 to 1.0, arranged in a gradient
    // so that top left is 0.0, bottom right is 1.0
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int x = i % width;
    int y = i / width;

    // Get normalised Manhattan distance
    float dist = (x + y)/(float)(width + height - 2);

    // Set the pixel value, with the peak being at the centre
    image[i] = dist;
}

__global__
void cudaCopyToSurface_kernel( hipSurfaceObject_t dest, float *src, int width )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int x = i % width;
    int y = i / width;

    surf2Dwrite( src[i], dest, x*sizeof(float), y );
}

__global__
void cudaRotatePoints_kernel( float *points, float rad )
{
    // Assumes "points" is an array of sets of (x,y) coords
    // (i.e. two floats per point), with stride 4
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = 4;
    float x = points[stride*i];
    float y = points[stride*i+1];
    float s, c;
    sincosf( rad, &s, &c );
    points[stride*i]   = c*x - s*y;
    points[stride*i+1] = s*x + c*y;
}

/*  ^      ^     ^
    |      |     |
   DEVICE FUNCTIONS
   ----------------
   HOST FUNCTIONS
    |     |     |
    v     v     v
*/

void cudaVDIFToFloatComplex( void *d_dest, void *d_src, size_t framelength, size_t headerlength, size_t NsNp, int c)
{
    dim3 blocks((NsNp-1)/1024+1);
    dim3 threads(1024);
    cudaVDIFToFloatComplex_kernel<<<blocks, threads>>>(
                (uint8_t *)d_src,
                (hipFloatComplex *)d_dest,
                framelength,
                headerlength,
                c);
}

void cudaStokesI( float *d_dest, hipFloatComplex *d_src, size_t nDualPolSamples )
{
    dim3 blocks((nDualPolSamples-1)/1024+1);
    dim3 threads(1024);
    cudaStokesI_kernel<<<blocks, threads>>>( d_src, d_dest );
}

void cudaRotatePoints( float *d_points, float rad )
{
    cudaRotatePoints_kernel<<<1,4>>>( d_points, rad );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}

void cudaCopyToSurface( hipSurfaceObject_t surf, float *d_image, int w, int h )
{
    cudaCopyToSurface_kernel<<<w,h>>>( surf, d_image, w );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}


float *cudaCreateImage( float *d_image, int w, int h )
{
    cudaCreateImage_kernel<<<w,h>>>( d_image, w, h );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    return d_image;
}

