#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "../src/cudaErrorChecking.h"

#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include <stdlib.h>
#include <stdio.h>

#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>

#include <cuda_gl_interop.h>

// GLUT-related constants
#define OPEN_FILE  1

// Mouse states
static double xprev;
static double yprev;
static bool drag_mode;

// Window states
static float windowWidth;
static float windowHeight;

#define XNORM(xpos)  ( (xpos)/windowWidth - 0.5)
#define YNORM(ypos)  (-(ypos)/windowHeight + 0.5)

static struct hipGraphicsResource *cudaPointsResource;
float *d_points;

static struct hipGraphicsResource *cudaImageResource;
texture<float, 2, hipReadModeElementType> d_image;
hipSurfaceObject_t surf;
hipResourceDesc surfRes;
hipArray *cuArray;
hipChannelFormatDesc channelDesc;

int w, h;

/**
 * Convert a VDIF buffer into an array of floats
 *
 * @param in              Pointer to the VDIF buffer
 * @param out             Pointer to the output buffer
 * @param frameSizeBytes  The number of bytes per VDIF frame
 *                        (including the header)
 * @param headerSizeBytes The number of bytes per VDIF frame header
 */
__global__ void cudaVDIFToFloatComplex( char2 *in, hipFloatComplex *out, int frameSizeBytes, int headerSizeBytes )
{
    // The size of just the data part of the frame
    int dataSizeBytes = frameSizeBytes - headerSizeBytes;

    // It is assumed that in points to the first byte in a frameheader
    int i = threadIdx.x + blockIdx.x*blockDim.x; // Index of (non-header) data sample

    // Express the index in terms of bytes
    int i2 = i*sizeof(char2);

    // Get the frame number for this byte, and the idx within this frame
    int frame      = i2 / dataSizeBytes;
    int idxInFrame = i2 % dataSizeBytes;

    // Calculate the indices into the input and output arrays for this sample
    int in_idx  = frame*frameSizeBytes + (headerSizeBytes + idxInFrame);
    int out_idx = i;

    // Bring the sample to register memory
    char2 sample = in[in_idx];

    // Turn it into a float and write it to global memory
    out[out_idx] = make_hipFloatComplex( (float)sample.x - 128.0, sample.y - 128.0 );
}

/**
 * Apply a phase ramp to complex data
 *
 * @param data          The data to which the phase ramp is applied (in-place)
 * @param radPerBin     The slope of the phase ramp (in radians per bin)
 * @param samplesPerBin The number of contiguous samples to be rotated by the
 *                      same amount
 */
__global__ void cudaApplyPhaseRamp( hipFloatComplex *data, float radPerBin, int samplesPerBin )
{
    // For this block/thread...
    int s = threadIdx.x + blockIdx.x*blockDim.x; // Get the (s)ample number
    int b = s / samplesPerBin;                   // Get the (b)in number

    // For each bin, calculate the phase rotation to be applied
    float rad = b * radPerBin;
    hipFloatComplex phase;
    sincosf( rad, &phase.y, &phase.x );

    // Apply the phase ramp (in-place)
    data[s] = hipCmulf( data[s], phase );
}

/**
 * Convert dual polarisation data to Stokes I
 *
 * @param data The data to be converted
 * @param stokesI The Stokes I output
 *
 * `data` is expected to be an array of *pairs* of complex numbers,
 * X,Y,X,Y,X,Y,...
 * from which the Stokes parameters are formed:
 *    I = |X|^2 + |Y|^2
 */
__global__ void cudaStokesI( hipFloatComplex *data, float *stokesI )
{
    // Let i represent the output sample index
    int i = threadIdx.x + blockIdx.x*blockDim.x;

    // Pull out the two polarisations
    hipFloatComplex X = data[2*i];
    hipFloatComplex Y = data[2*i + 1];

    // Calculate Stokes I
    stokesI[i] = X.x*X.x + X.y*X.y + Y.x*Y.x + Y.y*Y.y;
}


/*
int glut_main( int argc, char **argv )
{
    // Use a single buffered window in RGB mode (as opposed to a double-buffered
    // window or color-index mode).
    glutInit( &argc, argv );
    glutInitDisplayMode( GLUT_SINGLE | GLUT_RGB );

    // Position window at (80,80)-(480,380) and give it a title.
    glutInitWindowPosition( 80, 80 );
    glutInitWindowSize( 400, 300 );
    glutCreateWindow( "A Simple Triangle" );

    // Tell GLUT that whenever the main window needs to be repainted that it
    // should call the function display().
    glutDisplayFunc( display );

    // Prepare some test data
    size_t nFrames            = 4;
    size_t frameSizeBytes     = 544;
    size_t headerSizeBytes    = 32;
    size_t dataSizeBytes      = frameSizeBytes - headerSizeBytes;
    size_t nSamples           = nFrames * dataSizeBytes / sizeof(char2);
    size_t nPols              = 2;
    size_t nDualPolSamples    = nSamples / nPols;
    size_t vdifSizeBytes      = frameSizeBytes*nFrames;
    size_t vdifDataSizeBytes = nSamples * sizeof(hipFloatComplex);
    size_t stokesISizeBytes   = nDualPolSamples * sizeof(float);

    // Allocate memory
    char2 *vdif, *d_vdif;
    hipFloatComplex *d_vdifData;
    float *d_StokesI;

    gpuErrchk( hipHostMalloc( &vdif, vdifSizeBytes ) );
    gpuErrchk( hipMalloc( &d_vdif, vdifSizeBytes ) );
    gpuErrchk( hipMalloc( &d_vdifData, vdifDataSizeBytes ) );
    gpuErrchk( hipMalloc( &d_StokesI, stokesISizeBytes ) );

    FILE *f = fopen( "../tests/testdata.vdif", "r" );
    fread( vdif, vdifSizeBytes, 1, f );
    fclose( f );

    // Load it up and strip the headers
    gpuErrchk( hipMemcpy( d_vdif, vdif, nFrames * frameSizeBytes, hipMemcpyHostToDevice ) );
    gpuErrchk( hipDeviceSynchronize() );

    cudaVDIFToFloatComplex<<<nSamples/1024, 1024>>>( d_vdif, d_vdifData, frameSizeBytes, headerSizeBytes );
    gpuErrchk( hipDeviceSynchronize() );

    cudaStokesI<<<nDualPolSamples/1024, 1024>>>( d_vdifData, d_StokesI );
    gpuErrchk( hipDeviceSynchronize() );

    // Tell GLUT to start reading and processing events.  This function
    // never returns; the program only exits when the user closes the main
    // window or kills the process.
    glutMainLoop();

    // The following is never reached!!
    // Clean up memory
    gpuErrchk( hipFree( d_vdif ) );
    gpuErrchk( hipFree( d_vdifData ) );
    gpuErrchk( hipFree( d_StokesI ) );
    gpuErrchk( hipHostFree( vdif ) );
}
*/

void mouse_button_callback( GLFWwindow *window, int button, int action, int mods )
{
    if (button == GLFW_MOUSE_BUTTON_LEFT)
    {
        size_t size;
        switch (action)
        {
            case GLFW_PRESS:
                glfwGetCursorPos( window, &xprev, &yprev );
                drag_mode = true;
                gpuErrchk( hipGraphicsMapResources( 1, &cudaPointsResource, 0 ) );
                gpuErrchk( hipGraphicsResourceGetMappedPointer( (void **)&d_points, &size, cudaPointsResource ) );
                break;
            case GLFW_RELEASE:
                drag_mode = false;
                gpuErrchk( hipGraphicsUnmapResources( 1, &cudaPointsResource, 0 ) );
                break;
        }
    }
}

__global__
void cudaCreateImage( hipSurfaceObject_t surf, int width, int height )
{
    // Makes an image of pixels ranging from 0.0 to 1.0, arranged in a gradient
    // so that top left is 0.0, bottom right is 1.0
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Get normalised Manhattan distance
    float dist = (x + y)/(float)(width + height - 2);

    // Set the pixel value, with the peak being at the centre
    surf2Dwrite( dist, surf, x*sizeof(float), y );
}

__global__
void cudaRotatePoints( float *points, float rad )
{
    // Assumes "points" is an array of sets of (x,y) coords
    // (i.e. two floats per point), with stride 4
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = 4;
    float x = points[stride*i];
    float y = points[stride*i+1];
    float s, c;
    sincosf( rad, &s, &c );
    points[stride*i]   = c*x - s*y;
    points[stride*i+1] = s*x + c*y;
}

__global__
void cudaChangeBrightness( float *image, float amount )
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    image[i] += amount;
}

void cursor_position_callback( GLFWwindow* window, double xpos, double ypos )
{
    if (drag_mode)
    {
        float rad = atan2(YNORM(ypos),  XNORM(xpos)) -
                    atan2(YNORM(yprev), XNORM(xprev));
        //float dy = YNORM(ypos) - YNORM(yprev);

        // OpenGL CUDA interoperability
        cudaRotatePoints<<<1,4>>>( d_points, rad );
        //cudaChangeBrightness<<<1,36>>>( d_image, dy );

        xprev = xpos;
        yprev = ypos;
    }
}

// This function allocates memory
char *loadFileContentsAsStr( const char *filename )
{
    // Open the file for reading
    FILE *f = fopen( filename, "r" );
    if (f == NULL)
    {
        fprintf( stderr, "error: loadFileContentsAsStr: unable to open file "
                "%s\n", filename );
        exit(EXIT_FAILURE);
    }

    // Get the size of the file
    fseek( f, 0L, SEEK_END );
    long size = ftell( f );
    rewind( f );

    // Allocate memory in a string buffer
    char *str = (char *)malloc( size + 1 );

    // Read in the file contents to the string buffer
    long nread = fread( str, 1, size, f );
    if (nread != size)
    {
        fprintf( stderr, "warning: loadFileContentsAsStr: reading in "
                "contents of %s truncated (%ld/%ld bytes read)\n",
                filename, nread, size );
    }

    // Put a null termination at the end
    str[size] = '\0';

    return str;
}

int main( int argc, char *argv[] )
{
    // Start GL context and O/S window using the GLFW helper library
    glfwInit();
    const char *glfwerr;
    int code = glfwGetError( &glfwerr );
    if (code != GLFW_NO_ERROR)
    {
        fprintf( stderr, "ERROR: could not start GLFW3: %s\n", glfwerr );
        return EXIT_FAILURE;
    }

    windowWidth = 640;
    windowHeight = 480;
    GLFWwindow* window = glfwCreateWindow( windowWidth, windowHeight, "DM Slider", NULL, NULL );
    if (!window)
    {
        fprintf(stderr, "ERROR: could not open window with GLFW3\n");
        glfwTerminate();
        return EXIT_FAILURE;
    }
    glfwMakeContextCurrent( window );

    // Set up mouse
    glfwSetMouseButtonCallback( window, mouse_button_callback );
    glfwSetCursorPosCallback( window, cursor_position_callback );
    drag_mode = false;

    // Start GLEW extension handler
    glewExperimental = GL_TRUE;
    glewInit();

    // Get version info
    const GLubyte* renderer = glGetString( GL_RENDERER ); // get renderer string
    const GLubyte* version = glGetString( GL_VERSION ); // version as a string
    printf( "Renderer: %s\n", renderer );
    printf( "OpenGL version supported %s\n", version );

    // Tell GL to only draw onto a pixel if the shape is closer to the viewer
    //glEnable( GL_DEPTH_TEST ); // enable depth-testing
    //glDepthFunc( GL_LESS ); // depth-testing interprets a smaller value as "closer"

    // Define some points (to make a square)
    float points[] = {
        // vertices   // texcoords
         0.5f,  0.5f, 1.0f, 1.0f,
         0.5f, -0.5f, 1.0f, 0.0f,
        -0.5f,  0.5f, 0.0f, 1.0f,
        -0.5f, -0.5f, 0.0f, 0.0f
    };

    // Define a place for the points to live in global memory
    //gpuErrchk( hipMalloc( (void **)&d_points, sizeof(points) ) );

    GLuint vbo = 0;
    glGenBuffers( 1, &vbo );
    glBindBuffer( GL_ARRAY_BUFFER, vbo );
    glBufferData( GL_ARRAY_BUFFER, 16 * sizeof(float), points, GL_STATIC_DRAW );

    // Prepare a resource for CUDA interoperability
    hipGraphicsGLRegisterBuffer( &cudaPointsResource, vbo, cudaGraphicsMapFlagsNone );

    GLuint vao = 0;
    glGenVertexArrays( 1, &vao );
    glBindVertexArray( vao );
    glBindBuffer( GL_ARRAY_BUFFER, vbo );

    glVertexAttribPointer( 0, 2, GL_FLOAT, GL_FALSE, 4*sizeof(float), NULL );
    glEnableVertexAttribArray( 0 );
    glVertexAttribPointer( 1, 2, GL_FLOAT, GL_FALSE, 4*sizeof(float), (void *)(2*sizeof(float)) );
    glEnableVertexAttribArray( 1 );

    // Texture
    GLuint tex;
    glGenTextures( 1, &tex );
    glBindTexture( GL_TEXTURE_2D, tex );

    glTexParameteri( GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT );
    glTexParameteri( GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT );
    glTexParameteri( GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST );
    glTexParameteri( GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST );

    /*
    float image[] = { 0.0f, 0.1f, 0.2f, 0.3f, 0.4f, 0.5f,
                      0.1f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f,
                      0.2f, 0.3f, 0.4f, 0.5f, 0.6f, 0.7f,
                      0.3f, 0.4f, 0.5f, 0.6f, 0.0f, 0.8f, // <--- deliberate black pixel
                      0.4f, 0.5f, 0.6f, 0.7f, 0.8f, 0.9f,
                      0.5f, 0.6f, 0.7f, 0.8f, 0.9f, 1.0f };
     */

    w = h = 8;
    glTexImage2D( GL_TEXTURE_2D, 0, GL_R32F, w, h, 0, GL_RED, GL_FLOAT, NULL );

    glBindTexture( GL_TEXTURE_2D, 0 );

    gpuErrchk(
            hipGraphicsGLRegisterImage(
                &cudaImageResource,
                tex,
                GL_TEXTURE_2D,
                hipGraphicsRegisterFlagsSurfaceLoadStore
                )
            );

    gpuErrchk( hipGraphicsMapResources( 1, &cudaImageResource, 0 ) );
    gpuErrchk( hipGraphicsSubResourceGetMappedArray( &cuArray, cudaImageResource, 0, 0 ) );

    // CUDA Surface
    memset( &surfRes, 0, sizeof(hipResourceDesc) );
    surfRes.resType = hipResourceTypeArray;
    surfRes.res.array.array = cuArray;
    gpuErrchk( hipCreateSurfaceObject( &surf, &surfRes ) );
    printf( "surfRes: (w, h) = %ld, %ld\n", surfRes.res.pitch2D.width, surfRes.res.pitch2D.height );
    //dim3 threads(w, h);
    dim3 threads(w, h);
    cudaCreateImage<<<1,threads>>>( surf, w, h );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipGraphicsUnmapResources( 1, &cudaImageResource, 0 ) );

    // Set up camera

    const char* vertex_shader   = loadFileContentsAsStr( "vert.shader" );
    const char* fragment_shader = loadFileContentsAsStr( "frag.shader" );

    GLuint vs = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vs, 1, &vertex_shader, NULL);
    glCompileShader(vs);
    GLuint fs = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fs, 1, &fragment_shader, NULL);
    glCompileShader(fs);

    GLuint shader_programme = glCreateProgram();
    glAttachShader(shader_programme, fs);
    glAttachShader(shader_programme, vs);
    glLinkProgram(shader_programme);

    // Set up camera
    glm::mat4 Model( 1.0f ), View( 1.0f );

    glUseProgram(shader_programme);

    GLint model = glGetUniformLocation( shader_programme, "Model" );
    glUniformMatrix4fv( model, 1, GL_FALSE, glm::value_ptr(Model) );
 
    GLint view = glGetUniformLocation( shader_programme, "View" );
    glUniformMatrix4fv( view, 1, GL_FALSE, glm::value_ptr(View) );
 
    while(!glfwWindowShouldClose(window))
    {
        // wipe the drawing surface clear
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        //glActiveTexture( GL_TEXTURE0 );
        glBindTexture( GL_TEXTURE_2D, tex );

        // draw points 0-3 from the currently bound VAO with current in-use shader
        glDrawArrays(GL_TRIANGLE_STRIP, 0, 4);
        // update other events like input handling
        glfwPollEvents();

        // put the stuff we've been drawing onto the display
        glfwSwapBuffers(window);
    }

    //gpuErrchk( hipFree( d_points ) );
    gpuErrchk( hipDestroySurfaceObject( surf ) );
    gpuErrchk( hipFreeArray( cuArray ) );

    // close GL context and any other GLFW resources
    glfwTerminate();

    return EXIT_SUCCESS;
}
