#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "../src/cudaErrorChecking.h"

#include <GL/gl.h>
#include <GL/glut.h>

#include <stdlib.h>
#include <stdio.h>

// GLUT-related constants
#define OPEN_FILE  1

/**
 * Convert a VDIF buffer into an array of floats
 *
 * @param in              Pointer to the VDIF buffer
 * @param out             Pointer to the output buffer
 * @param frameSizeBytes  The number of bytes per VDIF frame
 *                        (including the header)
 * @param headerSizeBytes The number of bytes per VDIF frame header
 */
__global__ void cudaVDIFToFloatComplex( char2 *in, hipFloatComplex *out, int frameSizeBytes, int headerSizeBytes )
{
    // The size of just the data part of the frame
    int dataSizeBytes = frameSizeBytes - headerSizeBytes;

    // It is assumed that in points to the first byte in a frameheader
    int i = threadIdx.x + blockIdx.x*blockDim.x; // Index of (non-header) data sample

    // Express the index in terms of bytes
    int i2 = i*sizeof(char2);

    // Get the frame number for this byte, and the idx within this frame
    int frame      = i2 / dataSizeBytes;
    int idxInFrame = i2 % dataSizeBytes;

    // Calculate the indices into the input and output arrays for this sample
    int in_idx  = frame*frameSizeBytes + (headerSizeBytes + idxInFrame);
    int out_idx = i;

    // Bring the sample to register memory
    char2 sample = in[in_idx];

    // Turn it into a float and write it to global memory
    out[out_idx] = make_hipFloatComplex( (float)sample.x - 128.0, sample.y - 128.0 );
}

/**
 * Apply a phase ramp to complex data
 *
 * @param data          The data to which the phase ramp is applied (in-place)
 * @param radPerBin     The slope of the phase ramp (in radians per bin)
 * @param samplesPerBin The number of contiguous samples to be rotated by the
 *                      same amount
 */
__global__ void cudaApplyPhaseRamp( hipFloatComplex *data, float radPerBin, int samplesPerBin )
{
    // For this block/thread...
    int s = threadIdx.x + blockIdx.x*blockDim.x; // Get the (s)ample number
    int b = s / samplesPerBin;                   // Get the (b)in number

    // For each bin, calculate the phase rotation to be applied
    float rad = b * radPerBin;
    hipFloatComplex phase;
    sincosf( rad, &phase.y, &phase.x );

    // Apply the phase ramp (in-place)
    data[s] = hipCmulf( data[s], phase );
}

/**
 * Convert dual polarisation data to Stokes I
 *
 * @param data The data to be converted
 * @param stokesI The Stokes I output
 *
 * `data` is expected to be an array of *pairs* of complex numbers,
 * X,Y,X,Y,X,Y,...
 * from which the Stokes parameters are formed:
 *    I = |X|^2 + |Y|^2
 */
__global__ void cudaStokesI( hipFloatComplex *data, float *stokesI )
{
    // Let i represent the output sample index
    int i = threadIdx.x + blockIdx.x*blockDim.x;

    // Pull out the two polarisations
    hipFloatComplex X = data[2*i];
    hipFloatComplex Y = data[2*i + 1];

    // Calculate Stokes I
    stokesI[i] = X.x*X.x + X.y*X.y + Y.x*Y.x + Y.y*Y.y;
}

// Clears the current window and draws a triangle.
void display()
{
    // Set every pixel in the frame buffer to the current clear color.
    glClear(GL_COLOR_BUFFER_BIT);

    // Drawing is done by specifying a sequence of vertices.  The way these
    // vertices are connected (or not connected) depends on the argument to
    // glBegin.  GL_POLYGON constructs a filled polygon.
    glBegin(GL_POLYGON);
    {
        glColor3f(1, 0, 0); glVertex3f(-0.6, -0.75, 0.5);
        glColor3f(0, 1, 0); glVertex3f(0.6, -0.75, 0);
        glColor3f(0, 0, 1); glVertex3f(0, 0.75, 0);
    }
    glEnd();

    // Flush drawing command buffer to make drawing happen as soon as possible.
    glFlush();
}



int main( int argc, char **argv )
{
    // Use a single buffered window in RGB mode (as opposed to a double-buffered
    // window or color-index mode).
    glutInit( &argc, argv );
    glutInitDisplayMode( GLUT_SINGLE | GLUT_RGB );

    // Position window at (80,80)-(480,380) and give it a title.
    glutInitWindowPosition( 80, 80 );
    glutInitWindowSize( 400, 300 );
    glutCreateWindow( "A Simple Triangle" );

    // Tell GLUT that whenever the main window needs to be repainted that it
    // should call the function display().
    glutDisplayFunc( display );

    // Prepare some test data
    size_t nFrames            = 4;
    size_t frameSizeBytes     = 544;
    size_t headerSizeBytes    = 32;
    size_t dataSizeBytes      = frameSizeBytes - headerSizeBytes;
    size_t nSamples           = nFrames * dataSizeBytes / sizeof(char2);
    size_t nPols              = 2;
    size_t nDualPolSamples    = nSamples / nPols;
    size_t vdifSizeBytes      = frameSizeBytes*nFrames;
    size_t vdifDataSizeBytes = nSamples * sizeof(hipFloatComplex);
    size_t stokesISizeBytes   = nDualPolSamples * sizeof(float);

    // Allocate memory
    char2 *vdif, *d_vdif;
    hipFloatComplex *d_vdifData;
    float *d_StokesI;

    gpuErrchk( hipHostMalloc( &vdif, vdifSizeBytes ) );
    gpuErrchk( hipMalloc( &d_vdif, vdifSizeBytes ) );
    gpuErrchk( hipMalloc( &d_vdifData, vdifDataSizeBytes ) );
    gpuErrchk( hipMalloc( &d_StokesI, stokesISizeBytes ) );

    FILE *f = fopen( "../tests/testdata.vdif", "r" );
    fread( vdif, vdifSizeBytes, 1, f );
    fclose( f );

    // Load it up and strip the headers
    gpuErrchk( hipMemcpy( d_vdif, vdif, nFrames * frameSizeBytes, hipMemcpyHostToDevice ) );
    gpuErrchk( hipDeviceSynchronize() );

    cudaVDIFToFloatComplex<<<nSamples/1024, 1024>>>( d_vdif, d_vdifData, frameSizeBytes, headerSizeBytes );
    gpuErrchk( hipDeviceSynchronize() );

    cudaStokesI<<<nDualPolSamples/1024, 1024>>>( d_vdifData, d_StokesI );
    gpuErrchk( hipDeviceSynchronize() );

    // Tell GLUT to start reading and processing events.  This function
    // never returns; the program only exits when the user closes the main
    // window or kills the process.
    glutMainLoop();

    // The following is never reached!!
    // Clean up memory
    gpuErrchk( hipFree( d_vdif ) );
    gpuErrchk( hipFree( d_vdifData ) );
    gpuErrchk( hipFree( d_StokesI ) );
    gpuErrchk( hipHostFree( vdif ) );
}
