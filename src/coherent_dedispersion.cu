#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "../src/cudaErrorChecking.h"

//#include <GL/gl.h>
//#include <GL/glut.h>

#include <GL/glew.h>
#define GLFW_DLL
#include <GLFW/glfw3.h>

#include <stdlib.h>
#include <stdio.h>

// GLUT-related constants
#define OPEN_FILE  1

// Mouse states
static double xpos;
static double ypos;
static bool drag_mode;

// Window states
static double lview;
static double rview;
static float windowWidth;
static float windowHeight;

/**
 * Convert a VDIF buffer into an array of floats
 *
 * @param in              Pointer to the VDIF buffer
 * @param out             Pointer to the output buffer
 * @param frameSizeBytes  The number of bytes per VDIF frame
 *                        (including the header)
 * @param headerSizeBytes The number of bytes per VDIF frame header
 */
__global__ void cudaVDIFToFloatComplex( char2 *in, hipFloatComplex *out, int frameSizeBytes, int headerSizeBytes )
{
    // The size of just the data part of the frame
    int dataSizeBytes = frameSizeBytes - headerSizeBytes;

    // It is assumed that in points to the first byte in a frameheader
    int i = threadIdx.x + blockIdx.x*blockDim.x; // Index of (non-header) data sample

    // Express the index in terms of bytes
    int i2 = i*sizeof(char2);

    // Get the frame number for this byte, and the idx within this frame
    int frame      = i2 / dataSizeBytes;
    int idxInFrame = i2 % dataSizeBytes;

    // Calculate the indices into the input and output arrays for this sample
    int in_idx  = frame*frameSizeBytes + (headerSizeBytes + idxInFrame);
    int out_idx = i;

    // Bring the sample to register memory
    char2 sample = in[in_idx];

    // Turn it into a float and write it to global memory
    out[out_idx] = make_hipFloatComplex( (float)sample.x - 128.0, sample.y - 128.0 );
}

/**
 * Apply a phase ramp to complex data
 *
 * @param data          The data to which the phase ramp is applied (in-place)
 * @param radPerBin     The slope of the phase ramp (in radians per bin)
 * @param samplesPerBin The number of contiguous samples to be rotated by the
 *                      same amount
 */
__global__ void cudaApplyPhaseRamp( hipFloatComplex *data, float radPerBin, int samplesPerBin )
{
    // For this block/thread...
    int s = threadIdx.x + blockIdx.x*blockDim.x; // Get the (s)ample number
    int b = s / samplesPerBin;                   // Get the (b)in number

    // For each bin, calculate the phase rotation to be applied
    float rad = b * radPerBin;
    hipFloatComplex phase;
    sincosf( rad, &phase.y, &phase.x );

    // Apply the phase ramp (in-place)
    data[s] = hipCmulf( data[s], phase );
}

/**
 * Convert dual polarisation data to Stokes I
 *
 * @param data The data to be converted
 * @param stokesI The Stokes I output
 *
 * `data` is expected to be an array of *pairs* of complex numbers,
 * X,Y,X,Y,X,Y,...
 * from which the Stokes parameters are formed:
 *    I = |X|^2 + |Y|^2
 */
__global__ void cudaStokesI( hipFloatComplex *data, float *stokesI )
{
    // Let i represent the output sample index
    int i = threadIdx.x + blockIdx.x*blockDim.x;

    // Pull out the two polarisations
    hipFloatComplex X = data[2*i];
    hipFloatComplex Y = data[2*i + 1];

    // Calculate Stokes I
    stokesI[i] = X.x*X.x + X.y*X.y + Y.x*Y.x + Y.y*Y.y;
}


/*
int glut_main( int argc, char **argv )
{
    // Use a single buffered window in RGB mode (as opposed to a double-buffered
    // window or color-index mode).
    glutInit( &argc, argv );
    glutInitDisplayMode( GLUT_SINGLE | GLUT_RGB );

    // Position window at (80,80)-(480,380) and give it a title.
    glutInitWindowPosition( 80, 80 );
    glutInitWindowSize( 400, 300 );
    glutCreateWindow( "A Simple Triangle" );

    // Tell GLUT that whenever the main window needs to be repainted that it
    // should call the function display().
    glutDisplayFunc( display );

    // Prepare some test data
    size_t nFrames            = 4;
    size_t frameSizeBytes     = 544;
    size_t headerSizeBytes    = 32;
    size_t dataSizeBytes      = frameSizeBytes - headerSizeBytes;
    size_t nSamples           = nFrames * dataSizeBytes / sizeof(char2);
    size_t nPols              = 2;
    size_t nDualPolSamples    = nSamples / nPols;
    size_t vdifSizeBytes      = frameSizeBytes*nFrames;
    size_t vdifDataSizeBytes = nSamples * sizeof(hipFloatComplex);
    size_t stokesISizeBytes   = nDualPolSamples * sizeof(float);

    // Allocate memory
    char2 *vdif, *d_vdif;
    hipFloatComplex *d_vdifData;
    float *d_StokesI;

    gpuErrchk( hipHostMalloc( &vdif, vdifSizeBytes ) );
    gpuErrchk( hipMalloc( &d_vdif, vdifSizeBytes ) );
    gpuErrchk( hipMalloc( &d_vdifData, vdifDataSizeBytes ) );
    gpuErrchk( hipMalloc( &d_StokesI, stokesISizeBytes ) );

    FILE *f = fopen( "../tests/testdata.vdif", "r" );
    fread( vdif, vdifSizeBytes, 1, f );
    fclose( f );

    // Load it up and strip the headers
    gpuErrchk( hipMemcpy( d_vdif, vdif, nFrames * frameSizeBytes, hipMemcpyHostToDevice ) );
    gpuErrchk( hipDeviceSynchronize() );

    cudaVDIFToFloatComplex<<<nSamples/1024, 1024>>>( d_vdif, d_vdifData, frameSizeBytes, headerSizeBytes );
    gpuErrchk( hipDeviceSynchronize() );

    cudaStokesI<<<nDualPolSamples/1024, 1024>>>( d_vdifData, d_StokesI );
    gpuErrchk( hipDeviceSynchronize() );

    // Tell GLUT to start reading and processing events.  This function
    // never returns; the program only exits when the user closes the main
    // window or kills the process.
    glutMainLoop();

    // The following is never reached!!
    // Clean up memory
    gpuErrchk( hipFree( d_vdif ) );
    gpuErrchk( hipFree( d_vdifData ) );
    gpuErrchk( hipFree( d_StokesI ) );
    gpuErrchk( hipHostFree( vdif ) );
}
*/

void mouse_button_callback( GLFWwindow *window, int button, int action, int mods )
{
    if (button == GLFW_MOUSE_BUTTON_LEFT)
    {
        switch (action)
        {
            case GLFW_PRESS:
                glfwGetCursorPos( window, &xpos, &ypos );
                drag_mode = true;
                fprintf( stderr, "Clicked: (x, y) = (%lf, %lf)\n", xpos, ypos );
                break;
            case GLFW_RELEASE:
                glfwGetCursorPos( window, &xpos, &ypos );
                drag_mode = false;
                fprintf( stderr, "Released: (x, y) = (%lf, %lf)\n", xpos, ypos );
                break;
        }
    }
}

void cursor_position_callback( GLFWwindow* window, double xpos, double ypos )
{
    if (drag_mode)
    {
        fprintf( stderr, "Dragging: (x, y) = (%lf, %lf)\n", xpos, ypos );
    }
}

int main( int argc, char *argv[] )
{
    // Start GL context and O/S window using the GLFW helper library
    glfwInit();
    const char *glfwerr;
    int code = glfwGetError( &glfwerr );
    if (code != GLFW_NO_ERROR)
    {
        fprintf( stderr, "ERROR: could not start GLFW3: %s\n", glfwerr );
        return EXIT_FAILURE;
    }

    windowWidth = 640;
    windowHeight = 480;
    GLFWwindow* window = glfwCreateWindow( windowWidth, windowHeight, "DM Slider", NULL, NULL );
    if (!window)
    {
        fprintf(stderr, "ERROR: could not open window with GLFW3\n");
        glfwTerminate();
        return EXIT_FAILURE;
    }
    glfwMakeContextCurrent( window );

    // Set up mouse
    glfwSetMouseButtonCallback( window, mouse_button_callback );
    glfwSetCursorPosCallback( window, cursor_position_callback );
    drag_mode = false;

    // Start GLEW extension handler
    glewExperimental = GL_TRUE;
    glewInit();

    // Get version info
    const GLubyte* renderer = glGetString( GL_RENDERER ); // get renderer string
    const GLubyte* version = glGetString( GL_VERSION ); // version as a string
    printf( "Renderer: %s\n", renderer );
    printf( "OpenGL version supported %s\n", version );

    // Tell GL to only draw onto a pixel if the shape is closer to the viewer
    glEnable( GL_DEPTH_TEST ); // enable depth-testing
    glDepthFunc( GL_LESS ); // depth-testing interprets a smaller value as "closer"

    // Define a triangle
    float points[] = {
        0.5f,  0.5f,  0.0f,
        0.5f, -0.5f,  0.0f,
        -0.5f, 0.5f,  0.0f,
        -0.5f, -0.5f,  0.0f
    };

    GLuint vbo = 0;
    glGenBuffers( 1, &vbo );
    glBindBuffer( GL_ARRAY_BUFFER, vbo );
    glBufferData( GL_ARRAY_BUFFER, 12 * sizeof(float), points, GL_STATIC_DRAW );

    GLuint vao = 0;
    glGenVertexArrays( 1, &vao );
    glBindVertexArray( vao );
    glEnableVertexAttribArray( 0 );
    glBindBuffer( GL_ARRAY_BUFFER, vbo );
    glVertexAttribPointer( 0, 3, GL_FLOAT, GL_FALSE, 0, NULL );

    const char* vertex_shader =
        "#version 400\n"
        "in vec3 vp;"
        "void main() {"
        "  gl_Position = vec4(vp, 1.0);"
        "}";

    const char* fragment_shader =
        "#version 400\n"
        "out vec4 frag_colour;"
        "void main() {"
        "  frag_colour = vec4(0.5, 0.0, 0.5, 1.0);"
        "}";

    GLuint vs = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vs, 1, &vertex_shader, NULL);
    glCompileShader(vs);
    GLuint fs = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fs, 1, &fragment_shader, NULL);
    glCompileShader(fs);

    GLuint shader_programme = glCreateProgram();
    glAttachShader(shader_programme, fs);
    glAttachShader(shader_programme, vs);
    glLinkProgram(shader_programme);

    /*
    // At the moment, this doesn't seem to do anything
    // Set up view
    lview = -1.0;
    rview = 1.0;
    glMatrixMode( GL_PROJECTION );
    glLoadIdentity();
    //glOrtho( 0.0f, windowWidth, windowHeight, 0.0f, 0.0f, 1.0f );
    glOrtho( 0.0f, 1.0f, 1.0f, 0.0f, 0.0f, 1.0f );
    glMatrixMode( GL_MODELVIEW );
    */

    while(!glfwWindowShouldClose(window))
    {
        // wipe the drawing surface clear
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        glUseProgram(shader_programme);
        glBindVertexArray(vao);
        // draw points 0-3 from the currently bound VAO with current in-use shader
        glDrawArrays(GL_TRIANGLE_STRIP, 0, 4);
        // update other events like input handling
        glfwPollEvents();
        // put the stuff we've been drawing onto the display
        glfwSwapBuffers(window);
    }

    // close GL context and any other GLFW resources
    glfwTerminate();

    return EXIT_SUCCESS;
}
